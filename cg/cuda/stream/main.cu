#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernelOperation(float* d_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_data[idx] = d_data[idx] * 2.0f;
    }
}

int main() {
    int n = 1 << 20;
    size_t size = n * sizeof(float);

    // Allocate host memory
    float* h_data = (float*)malloc(size);

    for (int i = 0; i < n; i++) {
        h_data[i] = i * 0.5f;
    }

    // Allocate device memory
    float* d_data;
    hipMalloc(&d_data, size);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Async copy and kernel execution
    hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream);
    kernelOperation<<<(n + 255) / 256, 256, 0, stream>>>(d_data, n);
    hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    // Verify result
    for (int i = 0; i < 10; i++) {
        std::cout << h_data[i] << std::endl;
    }

    // Free memory
    free(h_data);
    hipFree(d_data);
    hipStreamDestroy(stream);

    return 0;
}
